#include <iostream>
#include <common.cuh>
#include "v1.h"

using datatype = float;

int main()
{
    constexpr int M = 1024, N = 16, K = 1024;
    constexpr int loop_time = 100;
    
    datatype *h_a, *h_b, *h_c;
    datatype *d_a, *d_b, *d_c;
    datatype alpha = 1.0f, beta = 0.0f;

    hipHostMalloc(&h_a, sizeof(datatype) * M * K);
    hipHostMalloc(&h_b, sizeof(datatype) * K * N);
    hipHostMalloc(&h_c, sizeof(datatype) * M * N);

    hipMalloc(&d_a, sizeof(datatype) * M * K);
    hipMalloc(&d_b, sizeof(datatype) * K * N);
    hipMalloc(&d_c, sizeof(datatype) * M * N);

    hipMemcpy(d_a, h_a, sizeof(datatype) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(datatype) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, sizeof(datatype) * M * N, hipMemcpyHostToDevice);
    for(int i = 0; i < loop_time; i ++) run_v1(M, N, K, d_a, d_b, d_c);

    ///////////////////////////////////////////////////////////////
    Timer t;
    t.start();
    for(int i = 0; i < loop_time; i ++)
        run_v1(M, N, K, d_a, d_b, d_c);
    t.stop();
    ///////////////////////////////////////////////////////////////

    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}