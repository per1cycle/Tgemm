#include <iostream>
#include <functional>
#include <common.cuh>
#include "v1.h"

using datatype = float;
using kernel_function = std::function<void(uint, uint, uint, float*, float*, float*)>;

// todo
std::vector<kernel_function> kernels;

int main()
{
    constexpr int M = 1024, N = 16, K = 1024;
    constexpr int loop_time = 1;
    
    datatype *h_a, *h_b, *h_c;
    datatype *d_a, *d_b, *d_c;
    datatype alpha = 1.0f, beta = 0.0f;

    hipHostMalloc(&h_a, sizeof(datatype) * M * K);
    hipHostMalloc(&h_b, sizeof(datatype) * K * N);
    hipHostMalloc(&h_c, sizeof(datatype) * M * N);

    hipMalloc(&d_a, sizeof(datatype) * M * K);
    hipMalloc(&d_b, sizeof(datatype) * K * N);
    hipMalloc(&d_c, sizeof(datatype) * M * N);

    hipMemcpy(d_a, h_a, sizeof(datatype) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(datatype) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, sizeof(datatype) * M * N, hipMemcpyHostToDevice);
    // check if the kernel give correct result.
    kernels.push_back(run_v1);
    ///////////////////////////////////////////////////////////////
    
    Timer t;
    for(int i = 0; i < kernels.size(); i ++)
    {
        const auto& kernel = kernels[i];
        t.start();

        for(int j = 0; j < loop_time; j ++)
        {
            kernel(M, N, K, d_a, d_b, d_c);
        }
        t.stop();
        t.report_sgemm_with_loop(M, N, K, alpha, beta, loop_time);
        t.reset();

    }

    ///////////////////////////////////////////////////////////////

    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
